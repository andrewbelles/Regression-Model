#include "../include/neural.hpp"
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <random>

int main(int argc, char *argv[]) {
  std::random_device rd; 
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> distribution(-5.0, 5.0);

  // Compute at compile time 
  constexpr int size = 256;
  constexpr dim3 blocks(16, 16);
  constexpr dim3 grid((size + blocks.x - 1) / blocks.x, (size + blocks.y) / blocks.y);

  // Generate Unified Memory for Matrices 
  Matrix *A = new_matrix(size, size); 
  Matrix *B = new_matrix(size, size);  

  // Vectors to fill matrices
  float *a_vec, *b_vec;

  // Allocate as unified
  hipMallocManaged(&a_vec, sizeof(float) * size);
  hipMallocManaged(&b_vec, sizeof(float) * size);

  // Fill with random values  
  for (int i = 0; i < size * size; i++) {
    a_vec[i] = distribution(gen);
    b_vec[i] = distribution(gen);
  }

  // Send data to gpu 
  hipMemPrefetchAsync(A, sizeof(Matrix), 0);
  hipMemPrefetchAsync(B, sizeof(Matrix), 0);
  hipMemPrefetchAsync(a_vec, sizeof(float) * size * size, 0);
  hipMemPrefetchAsync(b_vec, sizeof(float) * size * size, 0);
  hipDeviceSynchronize();

  // Fill matrices with data
  fill_matrix<<<grid, blocks>>>(A, a_vec);
  fill_matrix<<<grid, blocks>>>(B, b_vec);
  hipDeviceSynchronize();

  scale_matrix<<<grid, blocks>>>(B, 3.0); 
  A = transpose_matrix(A);

  hipDeviceSynchronize();
  Matrix *C = matrix_multiplication(A, B);

  // Chain complex operation for complexity
  A = transpose_matrix(A);
  matrix_elementwise_operation(B, A, Hadamard);
  A = transpose_matrix(A);
  Matrix *D = matrix_multiplication(A, B);
  matrix_elementwise_operation(C, D, Sub);

  // Prefetch back to cpu
  hipMemPrefetchAsync(C, sizeof(Matrix), hipCpuDeviceId);
  hipMemPrefetchAsync(C->data, sizeof(float) * C->rows() * C->cols(), hipCpuDeviceId);
  hipDeviceSynchronize();

  // print result 
  for (int i = 0; i < C->cols(); i++) {
    std::cout << "[";
    for (int j = 0; j < C->rows(); j++) {
      std::cout << " " << C->data[i * size + j]; 
    }
    std::cout << "]\n";
  }

  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(a_vec);
  hipFree(b_vec);

  return 0;
}
