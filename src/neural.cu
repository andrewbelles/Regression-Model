#include "hip/hip_runtime.h"
#include "../include/neural.hpp"
#include <hip/hip_runtime_api.h>

// #define __debug
// Neural Network Operations 

// Initialize the weights and biases for a layer depending on its activation function
__host__ static void initialize_layer(Layer *layer) {
  const uint col = layer->weights.cols();
  const uint row = layer->weights.rows();

  // Set grid/block sizes for kernel launch
  dim3 blocks(BLOCKSIZE, BLOCKSIZE);
  dim3 grid((col + BLOCKSIZE - 1) / BLOCKSIZE, (row + BLOCKSIZE - 1) / BLOCKSIZE);
  std::random_device rd;
  std::mt19937 gen(rd());
  float uniform_range = 0.0;
  float *weight_init, *bias_init;

  // Determine the type 
  switch (layer->type) {
    case ActivationType::Tanh:
    case ActivationType::Sigmoid:
    default:
      uniform_range = sqrtf(6.0 / static_cast<float>(col + row));
      break;
    case ActivationType::Leakyrelu:
    case ActivationType::Relu:
    case ActivationType::Elu:
      uniform_range = sqrtf(2.0 / static_cast<float>(row)); 
      break;
  }

  std::uniform_real_distribution<> distribution(-uniform_range, uniform_range);
  
  // Create vectors of initial values for weights and biases 
  hipMallocManaged(&weight_init, sizeof(float) * row * col);
  hipMallocManaged(&bias_init, sizeof(float) * col);
  for (int i = 0; i < col; i++) {
    for (int j = 0; j < row; j++) {
      weight_init[i * row + j] = distribution(gen); 
      bias_init[i] = 1e-3;
    }
  }

  hipMemPrefetchAsync(weight_init, sizeof(float) * row * col, 0);
  hipMemPrefetchAsync(bias_init, sizeof(float) * col, 0);
  hipDeviceSynchronize();

  // Fill matrices with initializing values
  fill_matrix<<<grid, blocks>>>(&layer->weights, weight_init);
  fill_matrix<<<grid, blocks>>>(&layer->biases, bias_init);
  hipDeviceSynchronize();
}

// Layer sizes is one longer than layer count 
__host__ static size_t calculate_network_size(uint *layer_sizes, uint layer_count, uint input_size) {
   
  // Find size of metadata
  uint64_t total_size = sizeof(Network);            // Network metadata
  total_size += sizeof(Layer) * layer_count;        // Layer metadata 
  total_size += sizeof(uint) * (layer_count + 1);   // Sizes array 
  total_size += sizeof(Matrix) * (layer_count); // Activation array metadata

  // Iterate over each discrete layer
  for (uint i = 0; i < layer_count; i++) {
    // Find current and previous neuron counts from array
    uint previous_size = layer_sizes[i];
    uint current_size  = layer_sizes[i + 1];

    // Calculate memory for weights and biases ( and each activation )
    uint64_t weights_data    = (previous_size * current_size) * sizeof(float);
    uint64_t bias_data       = (current_size) * sizeof(float);
    uint64_t activation_data = (input_size * current_size) *sizeof(float); 

    // Sum
    total_size += activation_data + weights_data + bias_data; 
  }

  return total_size;
}

// Takes array of sizes and array of Activation functions 
__host__ Network *new_network(
  uint *layer_sizes,
  uint layer_count,
  uint input_size,
  std::vector<ActivationType> types
) {
  Network *network;
  const uint64_t total_size = calculate_network_size(layer_sizes, layer_count, input_size);
  hipMallocManaged(&network, total_size);

  // Reinterpret initial cast to grab pointers to metadata
  network->layers      = reinterpret_cast<Layer*>(network + 1);
  network->activations = reinterpret_cast<Matrix*>(network->layers + layer_count);
  network->sizes       = reinterpret_cast<int*>(network->activations + layer_count + 1);
  network->layer_count = layer_count; 
  network->total_size  = total_size;
  
  for (int i = 0; i <= layer_count; i++) {
    network->sizes[i] = layer_sizes[i];
  }
  
  // Cast address into uint64_t type 
  uint64_t pointer_offset = (uint64_t)(network->sizes + layer_count + 1);

  // Provide ownership to pointers for each Layer
  for (int i = 0; i < layer_count; i++) {
    const uint previous_size = layer_sizes[i];
    const uint current_size  = layer_sizes[i + 1];

    // Set weights metadata and collect location of data pointer from offset 
    network->layers[i].weights.row  = previous_size;
    network->layers[i].weights.col  = current_size; 
    network->layers[i].weights.data = (float*)pointer_offset;
    pointer_offset += static_cast<uint64_t>(sizeof(float) * previous_size * current_size);

    // Set bias metadata and collect location of data pointer from offset 
    network->layers[i].biases.row   = 1;
    network->layers[i].biases.col   = current_size;
    network->layers[i].biases.data  = (float*)pointer_offset;
    pointer_offset += static_cast<uint64_t>(sizeof(float) * current_size);

    // Set layers function 
    network->layers[i].type = types[i];
    
    initialize_layer(&network->layers[i]);
  }

  // Pointer offset is onto activations now so the array's metadata can be set 
  for (int i = 0; i < layer_count; i++) {
    uint current_size = layer_sizes[i];

    network->activations[i].row  = current_size;
    network->activations[i].col  = input_size;
    network->activations[i].data = (float*)pointer_offset;
    pointer_offset += static_cast<uint64_t>(sizeof(float) * current_size * input_size);
  }

  // Send network to GPU. It should not return till free
  hipError_t err = hipMemPrefetchAsync(network, total_size, 0);
  if (err != hipSuccess) {
    std::cerr << "Prefetch Error: " << hipGetErrorString(err) << '\n';
    exit(EXIT_FAILURE);
  }
  return network;
}

// Efficient copy of memory from one source vector into N batch matrices
// 3D launch configuration to allow for only one kernel call
__global__ static void slice_input_vector(
  Matrix *dests,
  float *src,
  uint batch_count,
  uint batch_size,
  uint feature_count
) {
  const uint block = blockIdx.z;
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (block >= batch_count) return;

  // Pull batch for block thread is operating on 
  Matrix *dest = &dests[block];

  // Check in bounds and copy data 
  if (x < dest->cols() && y < dest->rows()) {
    uint src_index = block * batch_size * feature_count + y * feature_count + x;
    dest->data[y * dest->cols() + x] = src[src_index];
  }
}

// Takes some input vector and converts to N batch array of data split into sections. Forward pass will be a kernel operating on each batch(?)
// Allocate on arena? I don't think so(?) I kind of want activations to be the compilation of batches in a nice shared memory type layout 
__host__ Matrix *input_to_batch_array(
  ArenaAllocator &arena,
  float *input_vector,
  uint64_t input_size,
  uint feature_count,
  uint *batch_count
) {
  const uint BATCHSIZE = 64; 
  Matrix *batch_matrix;
  arena.reset();  // Force a reset to arena to ensure enough space 

  // Allocate memory on arena
  float *d_inputs = static_cast<float*>(arena.allocate(input_size * sizeof(float)));
  // Async call to copy data to input vector 
  hipMemcpyAsync(d_inputs, input_vector, input_size * sizeof(float), hipMemcpyHostToDevice, arena.get_stream());

  // Memory of input_matrix == batch_matrix 
  // Compute memory cost for array of matrices

  // Quotient + 1 will be allocated.
  uint quotient = input_size / BATCHSIZE;
  uint rem      = input_size % BATCHSIZE; 
  quotient += (rem != 0) ? 1 : 0;

  // Matrices will be BATCHSIZE x feature_count 
  // Allocate data arrays contiguous to matrix array 
  // Rem is just the row count of the batch, it'll be allocated to 32 byte aligned for simplicity
  uint64_t matrix_size = sizeof(Matrix);
  uint64_t matrix_data = (BATCHSIZE * feature_count) * sizeof(float);
  matrix_data = (matrix_data + 31) & ~31; // Ensure 32 byte aligned 
  uint64_t total_size  = quotient * (matrix_data + matrix_size);

  // Allocate entire block
  void *full_ptr;
  hipMallocManaged(&full_ptr, total_size);

  assert(reinterpret_cast<uint64_t>(full_ptr) % 32 == 0);

  // Fetch start 
  batch_matrix = static_cast<Matrix*>(full_ptr);

  for (int i = 0; i < quotient; i++) {
    // Get offsets and pointer arithemetic/cast pointer to owner
    uint64_t matrix_offset = i * (matrix_size + matrix_data);
    Matrix *current_matrix = reinterpret_cast<Matrix*>(static_cast<char*>(full_ptr) + matrix_offset);
    uint64_t data_offset   = matrix_offset + matrix_size;
    float *current_data    = reinterpret_cast<float*>(static_cast<char*>(full_ptr) + data_offset);

    assert(reinterpret_cast<uint64_t>(current_matrix) % 32 == 0);
    assert(reinterpret_cast<uint64_t>(current_data) % 32 == 0);

    // Copy metadata into current matrix 
    current_matrix->row  = (rem != 0 && i == quotient - 1) ? rem : BATCHSIZE;
    current_matrix->col  = feature_count;
    current_matrix->data = current_data;

    // Copy current matrix into batch array
    batch_matrix[i] = *current_matrix;
  }

  // Fill calls
  dim3 blocks(BLOCKSIZE, BLOCKSIZE);
  dim3 grid;

  // Ensure memory has been copied 
  hipDeviceSynchronize();

  // Array slice loop 
  grid = dim3((feature_count + BLOCKSIZE - 1) / BLOCKSIZE, (BATCHSIZE + BLOCKSIZE - 1) / BLOCKSIZE, quotient);
  slice_input_vector<<<grid, blocks, 0, arena.get_stream()>>>(batch_matrix, d_inputs, quotient, BATCHSIZE, feature_count);

  // Reset arena and prefetch the batch matrices to the gpu.
  arena.reset();
  hipMemPrefetchAsync(batch_matrix, total_size, 0);

  // Return batch array and update count
  *batch_count = quotient;
  return batch_matrix;
}

__global__ void insert_output(Matrix *activation, Matrix *d_output, uint batch, uint batch_size) {
  // Place Array into activation at specified location 
  const uint row = activation->rows(), col = activation->cols();
  const uint out_row = d_output->rows(), out_col = d_output->cols();
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (x < out_row && y < out_col) {
    activation->data[batch * (row * col) + x * row + y] = d_output->data[x * row + y];
  }
}

// We don't want d_outputs until an output is to be read.
// d_outputs is structured identical to u_batches but is allocated on arena 
__host__ void forward_propagation(
  Network *network,
  Matrix *u_batches,
  ArenaAllocator &arena,
  uint batch_count,
  uint input_size
) {
  const int *sizes = network->get_sizes();
  const uint layer_count = network->get_layer();
  dim3 blocks(BLOCKSIZE, BLOCKSIZE), grid;

  for (uint batch = 0; batch < batch_count; batch++) {
    uint current_row = u_batches[batch].rows();
    Matrix *d_output = nullptr;
    
    hipMemPrefetchAsync(network, network->total_size, 0);
    for (uint i = 0; i < layer_count; i++) {

      std::cout << "Batch: " << batch << " Layer: " << i << '\n';
      std::cout << "Current Row: " << current_row << '\n';
      std::cout << "Size: " << sizes[i+1] << '\n';
      
      // Multiply by weights 
      Matrix *current_input = (i == 0) ? &u_batches[batch] : d_output;
      if (i != 0) assert(d_output != nullptr);

      std::cout << arena.get_remaining() / 1e6 << " MB remaining\n";
      d_output = matrix_multiplication(
        current_row,
        sizes[i],
        sizes[i],
        sizes[i+1],
        current_input,
        &network->layers[i].weights,
        arena
      );

      // Add biases
      matrix_elementwise_operation(
        current_row,
        sizes[i+1],
        network->layers[i].biases.rows(),
        network->layers[i].biases.cols(),
        d_output,
        &network->layers[i].biases,
        Add
      );

      grid = dim3((current_row + blocks.x - 1) / blocks.x, (sizes[i+1] + blocks.y - 1) / blocks.y);

      // Apply activation function to z
      if (i != layer_count - 1) {
        activate(current_row, sizes[i+1], d_output, network->layers[i].type, false);
        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess) {
          std::cerr << "Activate Error: " << hipGetErrorString(err) << '\n';
          exit(EXIT_FAILURE);
        }
        // Must be fully activated before insertion 
      }      

      // Append d_output to u_outputs
      insert_output<<<grid, blocks>>>(&network->activations[i + 1], d_output, batch, batch_count);
      hipDeviceSynchronize();
      // Can't be done async since we need d_output to not be overwritten by batch 
    }
    arena.reset();
  }
}
