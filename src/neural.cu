#include "hip/hip_runtime.h"
#include "../include/neural.hpp"
#include <cmath>

// #define __debug
// Neural Network Operations 

// Initialize the weights and biases for a layer depending on its activation function
__host__ static void initialize_layer(Layer *layer) {
  const uint col = layer->weights.cols();
  const uint row = layer->weights.rows();

  // Set grid/block sizes for kernel launch
  dim3 blocks(BLOCKSIZE, BLOCKSIZE);
  dim3 grid((col + BLOCKSIZE - 1) / BLOCKSIZE, (row + BLOCKSIZE - 1) / BLOCKSIZE);
  std::random_device rd;
  std::mt19937 gen(rd());
  float uniform_range = 0.0;
  float *weight_init, *bias_init;

  // Determine the type 
  switch (layer->type) {
    case ActivationType::Tanh:
    case ActivationType::Sigmoid:
    default:
      uniform_range = sqrtf(6.0 / static_cast<float>(col + row));
      break;
    case ActivationType::Leakyrelu:
    case ActivationType::Relu:
    case ActivationType::Elu:
      uniform_range = sqrtf(2.0 / static_cast<float>(row)); 
      break;
  }

  std::uniform_real_distribution<> distribution(-uniform_range, uniform_range);
  
  // Create vectors of initial values for weights and biases 
  hipMallocManaged(&weight_init, sizeof(float) * row * col);
  hipMallocManaged(&bias_init, sizeof(float) * col);
  for (int i = 0; i < col; i++) {
    bias_init[i] = 0.0;
    for (int j = 0; j < row; j++) {
      weight_init[j * row + i] = distribution(gen); 
    }
  }

  hipMemPrefetchAsync(weight_init, sizeof(float) * row * col, 0);
  hipMemPrefetchAsync(bias_init, sizeof(float) * col, 0);
  hipDeviceSynchronize();

  // Fill matrices with initializing values
  fill_matrix<<<grid, blocks>>>(&layer->weights, weight_init);
  fill_matrix<<<grid, blocks>>>(&layer->biases, bias_init);
  hipDeviceSynchronize();
}

// Layer sizes is one longer than layer count 
__host__ static size_t calculate_network_size(uint *layer_sizes, uint layer_count, uint input_size) {
   
  // Find size of metadata
  uint64_t total_size = sizeof(Network);            // Network metadata
  total_size += sizeof(Layer) * layer_count;        // Layer metadata 
  total_size += sizeof(uint) * (layer_count + 1);   // Sizes array 
  total_size += sizeof(Matrix) * (layer_count + 1); // Activation array metadata

  total_size += (2 * sizeof(Matrix) * layer_count);

  // Iterate over each discrete layer
  for (uint i = 0; i < layer_count; i++) {
    // Find current and previous neuron counts from array
    uint previous_size = layer_sizes[i];
    uint current_size  = layer_sizes[i + 1];

    // Calculate memory for weights and biases ( and each activation )
    uint64_t weights_data    = (previous_size * current_size) * sizeof(float);
    uint64_t bias_data       = (current_size) * sizeof(float);
    uint64_t activation_data = (input_size * current_size) * sizeof(float); 

    // Sum
    total_size += (activation_data + weights_data + bias_data); 
  }

  total_size += (input_size * layer_sizes[0]) * sizeof(float);

  return total_size;
}

__global__ void print_matrix(Matrix *A) {
  const uint col = A->cols(), row = A->rows();
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y; 

  if (x < col && y < row) {
    const uint index = x * A->rows() + y;
    printf("(%u, %u): %f\n", x, y, A->data[index]);
  }
}

__host__ Network *new_network(
  uint *layer_sizes,
  uint layer_count,
  uint input_size,
  std::vector<ActivationType> types
) {
  Network *network; 
  const uint64_t total_size = calculate_network_size(layer_sizes, layer_count, input_size);
#ifdef __debug
  std::cout << "Layer_Count: " << layer_count << '\n';
#endif
  hipError_t err = hipMallocManaged(&network, total_size);
  if (err != hipSuccess) {
    std::cerr << "Network Malloc Failure: " << hipGetErrorString(err) << '\n';
    return nullptr;
  }
  
  network->total_size  = total_size;
  network->layer_count = layer_count;

  // Set pointers of struct manually 
  char* ptr = reinterpret_cast<char*>(network) + sizeof(Network);
  network->layers = reinterpret_cast<Layer*>(ptr);
  ptr += sizeof(Layer) * layer_count;
#ifdef __debug
  std::cout << "Layer Address: 0x" << std::hex << reinterpret_cast<uint64_t>(network->layers) << '\n';
#endif
  network->sizes = reinterpret_cast<uint*>(ptr);
  ptr += sizeof(uint) * (layer_count + 1);
#ifdef __debug  
  std::cout << "Sizes Address: 0x" << std::hex << reinterpret_cast<uint64_t>(network->sizes) << '\n';
#endif
  network->activations = reinterpret_cast<Matrix*>(ptr);
  ptr += sizeof(Matrix) * (layer_count + 1);
#ifdef __debug  
  std::cout << "Activations Address: 0x" << std::hex << reinterpret_cast<uint64_t>(network->activations) << '\n';
#endif
  for (int i = 0; i < layer_count; i++) {
    // Collect layer
    Layer& layer = network->layers[i];
    layer.weights.data = reinterpret_cast<float*>(ptr); 
    // Advance ptr 
    ptr += layer_sizes[i] * layer_sizes[i+1] * sizeof(float);

    layer.biases.data  = reinterpret_cast<float*>(ptr);
    ptr += layer_sizes[i+1] * sizeof(float);
  }

  for (int i = 0; i < layer_count; i++) {
    Layer& layer = network->layers[i];

    layer.type = types[i];
    layer.weights.row = layer_sizes[i];
    layer.weights.col = layer_sizes[i+1];
    layer.biases.row  = 1;
    layer.biases.col  = layer_sizes[i+1];
    initialize_layer(&layer);
  }

  for (int i = 0; i < layer_count + 1; i++) {
    Matrix& activation = network->activations[i];
    activation.row  = input_size;
    activation.col  = layer_sizes[i];
    activation.data = reinterpret_cast<float*>(ptr);
    ptr += (input_size * layer_sizes[i] * sizeof(float));
  }

  for (int i = 0; i < layer_count + 1; i++) {
    network->sizes[i] = layer_sizes[i];
  }

  return network;
}

__host__ Matrix *input_to_batch_array(
  ArenaAllocator& arena,
  float *input_vector,
  uint64_t input_size,
  uint feature_count,
  uint *batch_count
) {
  const uint batchsize = 64;
  Matrix *batches;
  arena.reset();  // Force arena reset 
  
  float *d_inputs = reinterpret_cast<float*>(arena.allocate(input_size * feature_count * sizeof(float)));
  hipError_t err = hipMemcpyAsync(d_inputs, input_vector, input_size * feature_count * sizeof(float), hipMemcpyHostToDevice, arena.get_stream());
  if (err != hipSuccess) {
    std::cerr << "Memcpy Failure: " << hipGetErrorString(err) << '\n';
    return nullptr;
  }

  // Calculate size of batch matrix 
  uint quotient  = input_size / batchsize; 
  uint remainder = input_size % batchsize;
  *batch_count = quotient + (remainder != 0 ? 1 : 0);

  uint64_t matrix_size = *batch_count * sizeof(Matrix);
  uint64_t data_size   = (quotient * batchsize + remainder) * feature_count * sizeof(float);
  uint64_t total_size  = matrix_size + data_size;

  err = hipMallocManaged(&batches, total_size);
  if (err != hipSuccess) {
    std::cerr << "Batches Malloc Error: " << hipGetErrorString(err) << '\n';
    return NULL;
  }

  char* ptr = reinterpret_cast<char*>(batches) + matrix_size;
  for (int i = 0; i < *batch_count; i++) {
    const uint rows = (i == *batch_count - 1 && remainder != 0) ? remainder : batchsize;

    batches[i].row  = rows;
    batches[i].col  = feature_count;
    batches[i].data = reinterpret_cast<float*>(ptr);
    ptr += (rows * feature_count * sizeof(float));
  }

  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cerr << "Sync Error: " << hipGetErrorString(err) << '\n';
    return nullptr;
  }
  
  // Shift d_inputs into each batch array
  hipStream_t stream = arena.get_stream();
  for (int i = 0; i < *batch_count; i++) {
    uint rows = batches[i].row;
    uint cols = feature_count * sizeof(float);
    
    float *src = d_inputs + i * batchsize * feature_count;
    float *dst = batches[i].data;

    // Copy 2d flattened array to batch
    err = hipMemcpy2DAsync(dst, cols, src, cols, cols, rows, hipMemcpyDeviceToDevice, stream);
  
    if (err != hipSuccess) {
      std::cerr <<"d_inputs to Batches Failure: " << hipGetErrorString(err) << '\n';
      return nullptr;
    }
  }

  hipStreamSynchronize(stream);


  arena.reset();

  return batches;
}


__global__ void insert_output(Matrix *activation, Matrix *d_output, uint batch, uint batch_size) {
  // Place Array into activation at specified location 
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;
  
  // Fill activation at correct index 
  if (x < d_output->rows() && y < d_output->cols()) {
    uint activation_start = batch * batch_size * activation->cols();
    uint activation_idx   = activation_start + x * activation->cols() + y;
    activation->data[activation_idx] = d_output->data[x * d_output->cols() + y];
  }
}

// We don't want d_outputs until an output is to be read.
// d_outputs is structured identical to u_batches but is allocated on arena 
__host__ void forward_propagation(
  Network *network,
  Matrix *u_batches,
  ArenaAllocator &arena,
  uint batch_count,
  uint input_size
) {
  const uint *sizes = network->get_sizes();
  const uint layer_count = network->get_layer();
  dim3 blocks(BLOCKSIZE, BLOCKSIZE), grid;
  const uint start_row = u_batches->rows(), start_col = u_batches->cols();

  hipError_t err = hipMemPrefetchAsync(network, network->total_size, 0);
  if (err != hipSuccess) {
    std::cerr << "Prefetch Error: " << hipGetErrorString(err) << '\n';
    return;
  }

  Matrix *d_output = nullptr;

  for (uint b = 0; b < batch_count; b++) {

    Matrix& batch = u_batches[b];
    
    uint current_row = batch.rows();
    assert(current_row == 64); 

    for (uint i = 0; i < layer_count; i++) {
      assert(network->layers[i].weights.data != nullptr);
      
      // Multiply by weights 
      Matrix *current_input = (i == 0) ? &batch : d_output;
      if (i != 0) assert(d_output != nullptr);

      d_output = matrix_multiplication(
        current_row,
        sizes[i],
        sizes[i],
        sizes[i+1],
        current_input,
        &network->layers[i].weights,
        arena
      );

      // Add biases
      matrix_elementwise_operation(
        current_row,
        sizes[i+1],
        network->layers[i].biases.rows(),
        network->layers[i].biases.cols(),
        d_output,
        &network->layers[i].biases,
        Add
      );

      grid = dim3((current_row + blocks.x - 1) / blocks.x, (sizes[i+1] + blocks.y - 1) / blocks.y);
      // std::cout << "{" << current_row << "}x{" << sizes[i+1] << "}\n";

      // Apply activation function to z
      if (i != layer_count - 1) {
        activate(current_row, sizes[i+1], d_output, network->layers[i].type, false);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
          std::cerr << "Activate Error: " << hipGetErrorString(err) << '\n';
          exit(EXIT_FAILURE);
        }
        // Must be fully activated before insertion 
      }      

      // Append d_output to u_outputs
      insert_output<<<grid, blocks>>>(&network->activations[i + 1], d_output, b, batch_count);
      err = hipDeviceSynchronize();
      if (err != hipSuccess) {
        std::cerr << "Insert Output Failure: " << hipGetErrorString(err) << '\n';
        return;
      }
    }
    arena.reset();
  }
}
